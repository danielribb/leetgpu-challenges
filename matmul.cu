#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row  < M and col < K){
            float sum = 0.0;
            for(int i = 0; i< N; i++){
                sum += A[row * N + i] * B[i * K + col];
            }
        C[row * K + col] = sum;
    }
}

void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = M * N * sizeof(float);
    size_t sizeB = N * K * sizeof(float);
    size_t sizeC = M * K * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    // Copy input data from host to device
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Launch the kernel
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
