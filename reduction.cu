#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void reduction(float *d_input, float *d_output, int N){
   extern __shared__ float sdata[];
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int tid = threadIdx.x;

   float x = (idx < N) ? d_input[idx] : 0.0f;
   sdata[tid] = x;
   __syncthreads();

   for(unsigned int s = blockDim.x/2; s > 0; s>>=1){
        if(tid < s){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
   }

   if(tid == 0){
        d_output[blockIdx.x] = sdata[0];
   }
}


void solve(const float* input, float* output, int N) {  
    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, input,  N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1)/blockDim.x);
    int shared_size = 256 * sizeof(float);
    reduction<<<gridDim, blockDim, shared_size>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
