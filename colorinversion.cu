#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i  < width * height  ){
     int pid = i * 4;
     image[pid] = 255 - image[pid];
     image[pid + 1] = 255 - image[pid + 1];
     image[pid + 2] = 255 - image[pid + 2];
    }
}

void solve(unsigned char* image, int width, int height) {
    unsigned char* d_image;
    int image_size = width * height * 4;

    // Allocate device memory
    hipMalloc(&d_image, image_size * sizeof(unsigned char));

    // Copy input data from host to device
    hipMemcpy(d_image, image, image_size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_image, width, height);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(image, d_image, image_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
}
